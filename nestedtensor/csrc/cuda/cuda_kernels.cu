#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2020 ByteDance Inc
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuda_kernels.h"
#include <assert.h>
#include <cstdio>
#include <cstdlib>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <hip/hip_fp16.h>

namespace nteffectivetransformer{

// gelu code from 
// https://github.com/NVIDIA/DeepLearningExamples/blob/master/FasterTransformer/v1/fastertransformer/cuda/cuda_kernels.cu#L26-L45
template <typename T>
__inline__ __device__
T gelu(T x)
{
  float cdf = 0.5f * 
    (1.0f + tanhf((0.7978845608028654f * (x + 0.044715f * x * x * x))));
  return x * cdf;
}

// reduce code from 
// https://github.com/NVIDIA/DeepLearningExamples/blob/master/FasterTransformer/v1/fastertransformer/cuda/cuda_kernels.cu#L47-L73

#define FINAL_MASK 0xffffffff

template <typename T>
__inline__ __device__
T warpReduceSum(T val)
{
  for(int mask = 16; mask > 0; mask >>= 1)
    val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
  return val;
}

template <typename T>
__inline__ __device__
T blockReduceSum(T val)
{
  static __shared__ T shared[32]; 
  int lane = threadIdx.x & 0x1f; 
  int wid = threadIdx.x >> 5;  

  val = warpReduceSum<T>(val);

  if(lane == 0)
    shared[wid] = val;
  __syncthreads();

  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : (T)0.0f;
  val = warpReduceSum(val);
  return val;
}

/// ***************************** add_bias + gelu *****************************

template <typename T>
__global__ 
void add_bias_act(T* out, const T* bias, int m, int n)
{
  T val, reg_bias;

  int row_id = blockIdx.x;
  int ite = n / blockDim.x;
  int tid = threadIdx.x;

  for(int i = 0; i < ite; ++i)
  {
    reg_bias = __ldg(&bias[i * blockDim.x + tid]);
    row_id = blockIdx.x;

    while(row_id < m){
      val = out[tid + i * blockDim.x + row_id * n]+ reg_bias;
      out[tid + i * blockDim.x + row_id * n] = gelu<T>(val);
      row_id += gridDim.x;
    }
  }
}

template <typename T>
void add_bias_act_kernelLauncher(
  T* out, const T* bias, int m, int n, hipStream_t stream)
{
  dim3 grid(max(m / 4, 1));
  dim3 block(n / 4);
  assert(block.x < 1024);
  add_bias_act<T><<<grid, block, 0, stream>>>(out, bias, m, n);
}

template void add_bias_act_kernelLauncher<float>(
  float* out, const float* bias, int m, int n, hipStream_t stream);

/// *********************************** fin ***********************************


/// ************************** add_bias + layer_norm **************************

template <typename T>
__global__ 
void add_bias_input_layernorm(
  T* out, const T* input, const T* bias, const T* gamma, 
  const T* beta, int m, int n)
{
  int tid = threadIdx.x;

  __shared__ float s_mean;
  __shared__ float s_variance;
  float mean =  0.0f;
  float variance = 0.0f;

  float local_out = 0.0f;
  for(int i = tid; i < n; i += blockDim.x)
    local_out += (float)(out[blockIdx.x * n + i] 
                    + input[blockIdx.x * n + i] + __ldg(&bias[i]));

  mean = blockReduceSum<float>(local_out);
  if(threadIdx.x == 0)
    s_mean = mean / n;
  __syncthreads();

  variance = blockReduceSum<float>((
      local_out - s_mean) * (local_out - s_mean));
  if(threadIdx.x == 0)
    s_variance = variance / n + 1e-6f;
  __syncthreads();

  for(int i = tid; i < n; i += blockDim.x)
    out[blockIdx.x * n + i] = 
        (T)(((local_out - s_mean) * rsqrtf(s_variance)) 
      * (float)(__ldg(&gamma[i])) + (float)(__ldg(&beta[i])));
}

template<typename T>
void add_bias_input_layernorm_kernelLauncher(
  T* out, const T* input, const T* bias, 
  const T* gamma, const T* beta, int m, int n, hipStream_t stream)
{
  assert(n < 1024);
  dim3 grid(m);
  dim3 block(n);
  add_bias_input_layernorm<T><<<grid, block, 0, stream>>>(
    out, input, bias, gamma, beta, m, n);
}

template void add_bias_input_layernorm_kernelLauncher<float>(
  float* out, const float* input, 
  const float* bias, const float* gamma, const float* beta, 
  int m, int n, hipStream_t stream);

/// *********************************** fin ***********************************


/// *********************** compresse transformer input ***********************

__global__ 
void compress_bert_input(
  // const T* from_tensor,
  const int* mask, const int* prefix_sum, 
  // T* to_tensor,
  int* batch_idx, int* word_idx,
  int batch_size , int seq_len, int hidden_dim) 
{
  int bid = blockIdx.y;  // batch
  int wid = blockIdx.x;  // word 
  int tid = threadIdx.x; // 
  
  /// 1. count pos for from tensor 
  int mask_idx  = bid * seq_len + wid;

  if (mask[mask_idx] > 0.5) {
    int valid_idx = prefix_sum[mask_idx];

    /// 2. wirte batch id and word id for each word
    if (tid == 0) {
      batch_idx[valid_idx] = bid;
      word_idx[valid_idx]  = wid;
    }
    
    // /// 3. copy src data
    // float* src_ptr = (float*)from_tensor;
    // float* dst_ptr = (float*)to_tensor;
    // int src_idx = mask_idx  * hidden_dim + tid;
    // int dst_idx = valid_idx * hidden_dim + tid;
    // dst_ptr[dst_idx] = src_ptr[src_idx];
  }
}

void compressBertInput_kernelLauncher(
    // const T* from_tensor,
    const int* mask, const int* prefix_sum, 
    // T* to_tensor,
    int* batch_idx, int* word_idx,
    int batch_size , int seq_len, int hidden_dim, hipStream_t stream) 
{
  /// TODO : fp32
  dim3 grid(seq_len, batch_size);
  dim3 block(hidden_dim);
  // dim3 block(1);
  assert(hidden_dim <= 1024);
  compress_bert_input<<<grid, block, 0, stream>>>(
    // from_tensor,
    mask, prefix_sum, 
    // to_tensor,
    batch_idx, word_idx,
    batch_size , seq_len, hidden_dim);
  return;
}

/// *********************************** fin ***********************************

/// *********************** restore transformer output ************************
template<typename T>
__global__
void restore_bert_output(
    T* to_tensor,
    const T* from_tensor, const int*  batch_idx, const int* word_idx, 
    int valid_word_num, int seq_len, int hidden_dim) 
{
  int bid = batch_idx[blockIdx.x];
  int wid = word_idx[blockIdx.x]; 
  int tid = threadIdx.x;
  int vid = blockIdx.x;

  /// 3. copy src data
  float* src_ptr = (float*)from_tensor;
  float* dst_ptr = (float*)to_tensor;
  int src_idx = vid * hidden_dim + tid;
  int dst_idx = (bid * seq_len + wid) * hidden_dim + tid;
  dst_ptr[dst_idx] = src_ptr[src_idx];
}

template<typename T>
void restoreBertOutput_kernelLauncher(
    T* to_tensor,
    const T* from_tensor, const int* batch_idx, const int* word_idx, 
    int valid_word_num, int seq_len, int hidden_dim, hipStream_t stream) 
{
  // TODO : fp32
  dim3 grid(valid_word_num);
  dim3 block(hidden_dim);
  assert(hidden_dim <= 1024);
  restore_bert_output<<<grid, block, 0, stream>>>(
    to_tensor, 
    from_tensor, batch_idx, word_idx,
    valid_word_num, seq_len, hidden_dim);
}

template void restoreBertOutput_kernelLauncher<float>(
  float* to_tensor,
  const float* from_tensor, const int*  batch_idx, const int* word_idx, 
  int valid_word_num, int seq_len, int hidden_dim, hipStream_t stream);
  
/// *********************************** fin ***********************************

/// ***************************** exclusive scan ******************************
// The scan code is rewritten based on this repo :
// https://github.com/mattdean1/cuda/tree/master/parallel-scan
// I only rewritted device memory allocation part.

int THREADS_PER_BLOCK = 512;
int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;
#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)

__global__ void prescan_large(int *output, const int *input, int n, int *sums) 
{
    extern __shared__ int temp[];

    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * n;
    
    int ai = threadID;
    int bi = threadID + (n / 2);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    temp[ai + bankOffsetA] = input[blockOffset + ai];
    temp[bi + bankOffsetB] = input[blockOffset + bi];

    int offset = 1;
    for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    __syncthreads();


    if (threadID == 0) { 
        sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
        temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
    } 
    
    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    output[blockOffset + ai] = temp[ai + bankOffsetA];
    output[blockOffset + bi] = temp[bi + bankOffsetB];
}

__global__ void prescan_arbitrary(
  int *output, const int *input, int n, int powerOfTwo)
{
    extern __shared__ int temp[];// allocated on invocation
    int threadID = threadIdx.x;

    int ai = threadID;
    int bi = threadID + (n / 2);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

    
    if (threadID < n) {
        temp[ai + bankOffsetA] = input[ai];
        temp[bi + bankOffsetB] = input[bi];
    }
    else {
        temp[ai + bankOffsetA] = 0;
        temp[bi + bankOffsetB] = 0;
    }
    

    int offset = 1;
  // build sum in place up the tree
    for (int d = powerOfTwo >> 1; d > 0; d >>= 1) 
    {
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (threadID == 0) {
    // clear the last element
        temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0; 
    }

    for (int d = 1; d < powerOfTwo; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    if (threadID < n) {
        output[ai] = temp[ai + bankOffsetA];
        output[bi] = temp[bi + bankOffsetB];
    }
}

__global__ void add(int *output, int length, int *n) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * length;

    output[blockOffset + threadID] += n[blockID];
}

__global__ void add(int *output, int length, const  int *n1, const int *n2) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * length;

    output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}

// from https://stackoverflow.com/a/12506181
int nextPowerOfTwo(int x) {
    int power = 1;
    while (power < x) {
        power *= 2;
    }
    return power;
}

void scanSmallDeviceArray(
  int *d_out, const int* d_in, const int length, const hipStream_t stream);
void scanLargeDeviceArray(
  int *d_out, const int* d_in, const int length, int *d_buf, 
  const hipStream_t stream);
void scanLargeEvenDeviceArray(
  int *d_out, const int* d_in, const int length, int *d_buf, 
  const hipStream_t stream);

void scanLargeEvenDeviceArray(
  int *d_out, const int* d_in, const int length, int *d_buf, 
  const hipStream_t stream) 
{
    const int blocks = length / ELEMENTS_PER_BLOCK;
    const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

  int *d_sums = d_buf;
  int *d_incr = d_buf + blocks;
    // hipMalloc((void **)&d_sums, blocks * sizeof(int));
    // hipMalloc((void **)&d_incr, blocks * sizeof(int));

    prescan_large<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize, stream>>>(
    d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);

    const int sumsArrThreadsNeeded = (blocks + 1) / 2;
    if (sumsArrThreadsNeeded > THREADS_PER_BLOCK) {
        // perform a large scan on the sums arr
        scanLargeDeviceArray(d_incr, d_sums, blocks, d_buf, stream);
    }
    else {
        // only need one block to scan sums arr so can use small scan
        scanSmallDeviceArray(d_incr, d_sums, blocks, stream);
    }

    add<<<blocks, ELEMENTS_PER_BLOCK, 0, stream>>>(
    d_out, ELEMENTS_PER_BLOCK, d_incr);
}

void scanSmallDeviceArray(
  int *d_out, const int* d_in, const int length, const hipStream_t stream) 
{
    int powerOfTwo = nextPowerOfTwo(length);
    prescan_arbitrary
    <<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int), stream >>>(
      d_out, d_in, length, powerOfTwo);
}

/// 
void scanLargeDeviceArray(
    int *d_out, const int* d_in, const int length, int *d_buf, 
    const hipStream_t stream) 
{
    int remainder = length % (ELEMENTS_PER_BLOCK);
    if (remainder == 0) {
        scanLargeEvenDeviceArray(d_out, d_in, length, d_buf, stream);
    }
    else {
        // perform a large scan on a compatible multiple of elements
        int lengthMultiple = length - remainder;
        scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple, d_buf, stream);

        // scan the remaining elements and add the (inclusive) 
    // last element of the large scan to this
        int *startOfOutputArray = &(d_out[lengthMultiple]);
        scanSmallDeviceArray(
      startOfOutputArray, &(d_in[lengthMultiple]), remainder, stream);

        add<<<1, remainder, 0, stream>>>(
      startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), 
      &(d_out[lengthMultiple - 1]));
    }
}

void exclusiveScan_kernelLauncher(
  int* d_out, const int* d_in, const int length, const hipStream_t stream) 
{
    if (length > ELEMENTS_PER_BLOCK) {
        scanLargeDeviceArray(d_out, d_in, length, d_out + length, stream);
    }
    else {
        scanSmallDeviceArray(d_out, d_in, length, stream);
    }
}

/// *********************************** fin ***********************************

}//namespace nteffectivetransformer
